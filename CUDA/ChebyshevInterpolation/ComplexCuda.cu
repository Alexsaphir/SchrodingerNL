#include "hip/hip_runtime.h"
#include "ComplexCuda.cuh"



//Operator overloading for cmplx

__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex z)
{
	double factor = exp(z.x);
	return make_hipDoubleComplex(factor * cos(z.y), factor * sin(z.y));
}


__device__ __host__ __inline__ cmplx operator+(const cmplx &a, const cmplx &b)
{
	return hipCadd(a, b);
}

__device__ __host__ __inline__ cmplx operator+(const double &a, const cmplx &b)
{
	return make_hipDoubleComplex(a + b.x, b.y);
}

__device__ __host__ __inline__ cmplx operator+(const cmplx  &a, const double &b)
{
	return make_hipDoubleComplex(a.x + b, a.y);
}

__device__ __host__ __inline__ cmplx operator-(const cmplx &a, const cmplx &b)
{
	return hipCsub(a, b);
}

__device__ __host__ __inline__ cmplx operator-(const double &a, const cmplx &b)
{
	return make_hipDoubleComplex(a - b.x, -b.y);
}

__device__ __host__ __inline__ cmplx operator-(const cmplx &a, const double &b)
{
	return make_hipDoubleComplex(a.x - b, a.y);
}

__device__ __host__ __inline__ cmplx operator-(const cmplx &a)
{
	return make_hipDoubleComplex(-a.x, -a.y);
}

__device__ __host__ __inline__ cmplx operator*(const cmplx &a, const cmplx &b)
{
	return hipCmul(a, b);
}

__device__ __host__ __inline__ cmplx operator*(const double &a, const cmplx &b)
{
	return make_hipDoubleComplex(a*hipCreal(b), a*hipCimag(b));
}

__device__ __host__ __inline__ cmplx operator*(const cmplx &a, const double &b)
{
	return make_hipDoubleComplex(b*hipCreal(a), b*hipCimag(a));
}

__device__ __host__ __inline__ cmplx operator/(const cmplx &a, const cmplx &b)
{
	return hipCdiv(a, b);
}

__device__ __host__ __inline__ cmplx operator/(const cmplx &a, const double &b)
{
	return make_hipDoubleComplex(hipCreal(a) / b, hipCimag(a) / b);
}

__device__ __host__ __inline__ cmplx operator/(const double &a, const cmplx &b)
{
	return make_hipDoubleComplex(a, 0) / b;
}

__device__ __host__ __inline__ cmplx iMul(const cmplx &a)
{
	return make_hipDoubleComplex(-hipCimag(a), hipCreal(a));
}

__device__ __host__ __inline__ cmplx iMul(const double &a)
{
	return make_hipDoubleComplex(0, a);
}

