#include "Grid.cuh"

Grid::Grid(double Xmin, double Xmax, int N) :m_X(Xmin, Xmax, N)
{
	m_nbPts = m_X.getN();
	m_h_V = new (std::nothrow) cmplx[m_nbPts];
	hipError_t Err = hipMalloc(&m_d_V, m_nbPts * sizeof(cmplx));
	if (Err != 0)
		throw std::exception("Memory Allocation Error!!!!");
}

Axis Grid::getAxis() const
{
	return m_X;
}

cmplx * Grid::getHostData() const
{
	return m_h_V;
}

cmplx * Grid::getDeviceData() const
{
	return m_d_V;
}

void Grid::syncDeviceToHost()
{
	hipMemcpy(m_h_V, m_d_V, m_nbPts * sizeof(cmplx), hipMemcpyDeviceToHost);
}

void Grid::syncHostToDevice()
{
	hipMemcpy(m_d_V, m_h_V, m_nbPts * sizeof(cmplx), hipMemcpyHostToDevice);
}

Grid::~Grid()
{
	delete[] m_h_V;
	hipFree(m_d_V);
}
