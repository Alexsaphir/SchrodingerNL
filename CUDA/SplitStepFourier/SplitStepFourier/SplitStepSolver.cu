#include "hip/hip_runtime.h"
#include "SplitStepSolver.cuh"

namespace
{
	__global__ void FFTResizekernel(cmplx * d_V, int nbPts)
	{
		int i = blockIdx.x *blockDim.x + threadIdx.x;
		if (i < nbPts)
		{
			d_V[i] = hipCmul(d_V[i], make_hipDoubleComplex(1. / static_cast<double>(nbPts), 0));
		}
	}

	void FFTResize(cmplx * d_V, int N)
	{
		FFTResizekernel << <KernelUtility::computeNumberOfBlocks(1024, N), 1024 >> > (d_V, N);
	}
}

namespace
{
	__global__ void LinearKernel(cmplx *Vf, double dt, double p, int N, double Length)
	{
		int i = blockIdx.x *blockDim.x + threadIdx.x;


		if (i < N)
		{
			cmplx k = make_hipDoubleComplex(0, 0);
			if (i < N / 2)
			{
				k = iMul(2.*M_PI *static_cast<double>(i) / Length);
				//k = iMul(2.*M_PI *static_cast<double>(i) / static_cast<double>(N));
			}
			if (i == N / 2)
			{
				k = make_hipDoubleComplex(0, 0);
			}
			if (i > N / 2)
			{
				k = iMul(2.*M_PI * (static_cast<double>(i - N)) / Length);
				//k = iMul(2.*M_PI * (static_cast<double>(i - N)) / static_cast<double>(N));
			}
			Vf[i] = cuCexp(iMul(p)*dt*.5*k*k)*Vf[i];

		}
	}

	__global__ void NLinearKernel(cmplx *V, double dt, double p, int N, double Length)
	{
		int i = blockIdx.x *blockDim.x + threadIdx.x;
		double kapa = 2.;
		if (i < N)
		{
			V[i] = cuCexp(iMul(dt)*p*kapa*V[i] * hipConj(V[i]))*V[i];
		}
	}

	void LinearStep(cmplx *Vf, double dt, double p, int N, double Length)
	{
		LinearKernel << <KernelUtility::computeNumberOfBlocks(1024, N), 1024 >> > (Vf, dt, p, N, Length);
	}

	void NonLinearStep(cmplx *V, double dt, double p, int N, double Length)
	{
		NLinearKernel << <KernelUtility::computeNumberOfBlocks(1024, N), 1024 >> >(V, dt, p, N, Length);
	}
}

namespace
{
	void phi1(cmplx * d_U, double dt, int N, double Length, hipfftHandle *plan)
	{
		//FFt
		hipfftExecZ2Z(*plan, d_U, d_U, HIPFFT_FORWARD);
		//L
		LinearStep(d_U, dt, 1., N, Length);
		//FFt-1
		hipfftExecZ2Z(*plan, d_U, d_U, HIPFFT_BACKWARD);
		FFTResize(d_U, N);

		//NL
		NonLinearStep(d_U, dt, 1., N, Length);

	}

	void phi2(cmplx * d_U, double dt, int N, double Length, hipfftHandle *plan)
	{
		//NL
		NonLinearStep(d_U, dt, .5, N, Length);
		
		//FFt
		hipfftExecZ2Z(*plan, d_U, d_U, HIPFFT_FORWARD);
		//L
		LinearStep(d_U, dt, 1., N, Length);
		//FFt-1
		hipfftExecZ2Z(*plan, d_U, d_U, HIPFFT_BACKWARD);
		FFTResize(d_U, N);

		//NL
		NonLinearStep(d_U, dt, .5, N, Length);

	}

	void phi4(cmplx * d_U, double dt, int N, double Length, hipfftHandle *plan)
	{
		double w = (2. + std::pow(2., 1. / 3.) + .5*std::pow(2., 2. / 3))/3.;
		phi2(d_U, w*dt, N, Length, plan);
		phi2(d_U, (1.-w)*dt, N, Length, plan);
		phi2(d_U, w*dt, N, Length, plan);
	}
}

void SplitStep(cmplx * d_U, double dt, int N, double Length, hipfftHandle *plan, int order)
{
	switch (order)
	{
	case 2: phi2(d_U, dt, N, Length, plan);
		break;
	case 4: phi4(d_U, dt, N, Length, plan);
		break;
	default:phi1(d_U, dt, N, Length, plan);
		break;
	}
}
