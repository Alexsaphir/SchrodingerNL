#include "hip/hip_runtime.h"
#include "SignalFFT.cuh"

namespace
{
	__global__ void kernelResizeDataFFT(cmplx * d_V, int nbPts)
	{
		int i = blockIdx.x *blockDim.x + threadIdx.x;
		if (i < nbPts)
		{
			d_V[i] = hipCmul(d_V[i], make_hipDoubleComplex(1. / sqrt(static_cast<double>(nbPts)), 0));
		}
	}

	__global__ void kernelFirstDerivative(cmplx * d_V, int nbPts)
	{
		int i = blockIdx.x *blockDim.x + threadIdx.x;

		double k = static_cast<double>(i);
		cmplx Freq;


		//Positive Frequency are between 0 and <N/2
		//Negative Frequency are between N/2and <N
		if (i < nbPts / 2)
			Freq = make_hipDoubleComplex(0, k *2.*M_PI*k / static_cast<double>(nbPts));
		else if (i > nbPts / 2)
			Freq = make_hipDoubleComplex(0, (k - static_cast<double>(nbPts))*2.*M_PI*k / static_cast<double>(nbPts));
		else
			Freq = make_hipDoubleComplex(0, 0);//k=N/2
		__syncthreads();
		if (i < nbPts)
			d_V[i] = hipCmul(Freq, d_V[i]);
	}

}

SignalFFT::SignalFFT(int nbPoints) :m_nbPts(nbPoints > 0 ? nbPoints : 0), m_d_V(NULL), m_h_V(NULL)
{
	//create plan for complex to complex
	hipfftPlan1d(&m_plan, nbPoints, HIPFFT_Z2Z, 1);
	hipHostMalloc(&m_h_V, m_nbPts * sizeof(cmplx));
	hipMalloc(&m_d_V, m_nbPts * sizeof(cmplx));

	m_thread = 1024;
	m_block = ((m_nbPts % m_thread) == 0) ? (m_nbPts / m_thread) : (1 + m_nbPts / m_thread);
	m_GPUOrder = true;
}

int SignalFFT::getSignalPoints() const
{
	return m_nbPts;
}

cmplx * SignalFFT::getHostData() const
{
	return m_h_V;
}

cmplx * SignalFFT::getDeviceData() const
{
	return m_d_V;
}

void SignalFFT::syncHostToDevice()
{
	hipMemcpy(m_d_V, m_h_V, m_nbPts * sizeof(cmplx), hipMemcpyHostToDevice);
}

void SignalFFT::syncDeviceToHost()
{
	hipMemcpy(m_h_V, m_d_V, m_nbPts * sizeof(cmplx), hipMemcpyDeviceToHost);
	m_GPUOrder = true;
}

void SignalFFT::computeFFT(Signal * src)
{
	m_GPUOrder = true;
	hipfftExecZ2Z(m_plan, src->getDeviceData() , m_d_V, HIPFFT_FORWARD);
	kernelResizeDataFFT << < m_block, m_thread >> > (m_d_V, m_nbPts);
}

void SignalFFT::ComputeSignal(Signal * dst)
{
	hipfftExecZ2Z(m_plan, m_d_V, dst->getDeviceData(), HIPFFT_BACKWARD);
	kernelResizeDataFFT << < m_block, m_thread >> > (dst->getDeviceData(), m_nbPts);
}

void SignalFFT::smoothFilterCesaro()
{
	reorderData();
	for (int k = -m_nbPts / 2; k < ( m_nbPts / 2); ++k)
	{
		m_h_V[k + m_nbPts / 2] = m_h_V[k + m_nbPts / 2] * (1. - std::abs(k) / (static_cast<double>(m_nbPts) / 2.));
	}
	cancelReorderData();
}

void SignalFFT::smoothFilterLanczos()
{
	reorderData();
	for (int k = -m_nbPts / 2; k < (m_nbPts / 2); ++k)
	{
		m_h_V[k + m_nbPts / 2] = m_h_V[k + m_nbPts / 2] * std::sin(2.*M_PI*static_cast<double>(k) / static_cast<double>(m_nbPts)) / (2.*M_PI*static_cast<double>(k) / static_cast<double>(m_nbPts));
	}
	cancelReorderData();
}

void SignalFFT::smoothFilterRaisedCosinus()
{
	reorderData();
	for (int k = -m_nbPts / 2; k < (m_nbPts / 2); ++k)
	{
		m_h_V[k + m_nbPts / 2] = m_h_V[k + m_nbPts / 2] * (.5 + .5*std::cos(2.*M_PI*static_cast<double>(k) / static_cast<double>(m_nbPts)));
	}
	cancelReorderData();
}

void SignalFFT::reorderData()
{
	if (!m_GPUOrder)
		return;
	else
	{
		for (int i = 0; i < m_nbPts / 2; ++i)
		{
			cmplx tmp = m_h_V[i];
			m_h_V[i] = m_h_V[i + m_nbPts / 2];
			m_h_V[i + m_nbPts / 2] = tmp;
		}
		m_GPUOrder = false;
	}
}

void SignalFFT::cancelReorderData()
{
	if (m_GPUOrder)
		return;
	else
	{
		for (int i = 0; i < m_nbPts / 2; ++i)
		{
			cmplx tmp = m_h_V[i];
			m_h_V[i] = m_h_V[i + m_nbPts / 2];
			m_h_V[i + m_nbPts / 2] = tmp;
		}
		m_GPUOrder = true;
	}
}

void SignalFFT::firstDerivative()
{
	kernelFirstDerivative << < m_block, m_thread >> > (getDeviceData(), m_nbPts);
}

SignalFFT::~SignalFFT()
{
	hipfftDestroy(m_plan);
	hipHostFree(m_h_V);
	hipFree(m_d_V);
}
