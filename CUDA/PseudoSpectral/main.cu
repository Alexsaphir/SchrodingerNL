#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

#include <hip/hip_runtime_api.h>

#include <iostream>
#include <fstream>
#include <string>

#include "Axis.h"
#include "Signal.cuh"
#include "SignalFFT.cuh"


//2*M_PI
#define M_PI2 6.2831853071795864769252867665590057683943387987502

#define N_FFT 512 //Frequency Sampling*Duration

void exportData(const Axis *X, const Signal *S, const std::string &name)
{
	std::ofstream file;
	file.open(name);
	file << 2 << std::endl;
	for (int i = 0; i < S->getSignalPoints(); ++i)
		file << X->getLinearValueAt(i) << " " << S->getHostData()[i].x << " " << S->getHostData()[i].y << "\n";//(S->getHostData()[i].x<0?-1.:1)*
	file.close();
}

void exportData(const Axis *X, SignalFFT *S, const std::string &name)
{//Export data of the host
	std::ofstream file;
	file.open(name);
	file << 1 << std::endl;
	for (int i = 0; i < S->getSignalPoints(); ++i)
		file << X->getFrequency(i) << " " << (S->getHostData()[i].x<0 ? -1. : 1.)*hipCabs(S->getHostData()[i]) << "\n";
	file.close();
}

void computeError(const Signal *S1, const Signal *S2, Signal *E)
{
	//S1, S2, E: same size
	for (int i = 0; i < S1->getSignalPoints(); ++i)
	{
		E->getHostData()[i] = S1->getHostData()[i] - S2->getHostData()[i];
	}
}

int main()
{
	Axis X(-1, 1, N_FFT);
	
	Signal S(-1, 1, N_FFT);//Input signal
	Signal Sout(-1, 1, N_FFT);//Output Signal

	SignalFFT Sfft(N_FFT);//FFT Signal

	GaussPulseLinear(&S, 5.);
	exportData(&X, &S, "Plot/data.ds");//Save the initial signal

	Sfft.computeFFT(&S);
	Sfft.syncDeviceToHost();//Send data to RAM
	Sfft.reorderData();//Shift Frequency for the data on the host
	exportData(&X, &Sfft, "Plot/dataFFT.ds");//Save FFT of the Signal computed

	//Sfft.smoothFilterCesaro();//Apply Filtering
	//Sfft.smoothFilterLanczos();
	Sfft.smoothFilterRaisedCosinus();

	Sfft.ComputeSignal(&Sout);//Get back the signal in physical space
	Sout.syncDeviceToHost();//Send data to RAM

	exportData(&X, &Sout, "Plot/dataN.ds");
	
	//getchar();
	return 0;
}
